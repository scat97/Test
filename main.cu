#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#define cimg_use_jpeg
#include "CImg.h"
#include <iostream>
#include "kernel.h"
#include "kernelcpu.h"
//#include "hip/hip_runtime_api.h"
#define BLOCKSIZE 128
#define BLOCKSIZE2 16
using namespace std;
using namespace cimg_library;


/////////////////
//surface<void, 2> surftest;

//////////

int compute_diff(unsigned char * res_cpu, unsigned char * res_gpu, unsigned long size){
  int res = 0;
  for(int i = 0;i < size; i++){
    res += res_cpu[i] - res_gpu[i];
  }
  return res;
}

int compute_diff_hist(int * res_cpu, int * res_gpu, unsigned long size){
    int res = 0;
    for(int i = 0;i < size; i++){
      res += res_cpu[i] - res_gpu[i];
    }
    return res;
  }


int main()
{
    //load image
    CImg<unsigned char> src("cat2.jpg"); // we use cat2.jpg to grade
    int width = src.width();
    int height = src.height();
    unsigned long size = src.size();
    unsigned long size2 = width*height;

    //create pointer to image
    unsigned char *h_src = src.data();
    
    CImg<unsigned char> dst(width, height, 1, 1);
    unsigned char *h_dst = dst.data();
    //Something contrast blabla
    CImg<unsigned char> contrast(width, height, 1, 1);
    unsigned char *h_contrast = contrast.data();

    // for contrast enhancemant
    CImg<unsigned char> smoothing_gpu(width, height, 1, 1);
    unsigned char *h_smoothing = smoothing_gpu.data();

    unsigned char *d_src;
    unsigned char *d_dst;
    unsigned char *d_dst_2;
    //hipMalloc(&d_dst_2, width*height*sizeof(unsigned int));

    unsigned char *GPU_contrast;
    unsigned char *GPU_smoothing;
    unsigned char *GPU_smoothing2;

    hipEvent_t start; // to record processing time
    hipEvent_t stop;
    float msecTotal,msecTotal2;
  


    std::cout << "Start CPU processing" << std::endl;
    // create and start timer

    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    unsigned char *cpu_ref = new unsigned char [width*height];
    int* cpu_ref_hist = new int[256];
    memset(cpu_ref_hist, 0, sizeof(int)*256);

    rgb2gray_cpu(h_src, cpu_ref, width, height);

    // hisgram cpu
    
    histgram_cpu(cpu_ref_hist, cpu_ref, width,  height);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    float cpu_time = msecTotal;

    std::cout <<"CPU processing time: " << cpu_time << " ms" <<std::endl;

    
    //std::cout << "Start GPU processing" << std::endl;
    // create and start timer
    //hipEventCreate(&start);
    //hipEventRecord(start, NULL); 

    hipMalloc((void**)&d_src, size);
    hipMalloc((void**)&d_dst, width*height*sizeof(unsigned char));

    hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice);

////////////////////////////////////////////////
/// RGB to gray ///

    //launch the kernel
    dim3 blkDim (BLOCKSIZE, 1,1);
    dim3 grdDim (ceil(size2/BLOCKSIZE), 1, 1);
    cout << (ceil(size2/BLOCKSIZE)) << " grid" << endl;
    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 

    rgb2gray<<<grdDim, blkDim>>>(d_src, d_dst, width, height);

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    cout << "RGB2Gray time GPU:" << msecTotal << "ms" << endl;

    hipDeviceSynchronize();
    hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);
    
//////////////////////////////////////////////////
    //for(int AmountOfHists = 16; AmountOfHists <= 16; AmountOfHists+= 4){
        int power = 2;
        int AmountOfHists = pow(2,power);
        int* hist = new int[256];
        int* histGPU = new int[256*AmountOfHists];
        // create and start timer
    ///////////////////////////////////////////////////////
        dim3 blkDim2 (256, 1,1);
        dim3 grdDim2 (ceil(size2/256),1, 1);
        hipEventCreate(&start);
        hipEventRecord(start, NULL); 

        hipMalloc(&histGPU, 256*sizeof(int)*AmountOfHists);
        hipMemset(histGPU, 0, 256*sizeof(int)*AmountOfHists);
        //cout << "malloc is " << temp3 << " memset is " << temp2 <<endl;
        int mask = pow(2,(power))-1;

        histgram<<<grdDim2,blkDim2>>>(histGPU, d_dst, width , mask);

        dim3 blkDim3 (256,1,1);
        for (int stride = AmountOfHists/2; stride>0; stride>>=1){ // Interleaved reduction
            dim3 grdDim3 (stride,1,1);
            histgram_summation<<<grdDim3, blkDim3>>>(histGPU, stride);
            hipDeviceSynchronize();
        }
  
        hipMemcpy(hist, histGPU, 256*sizeof(int),hipMemcpyDeviceToHost);
        
        hipEventCreate(&stop);
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&msecTotal, start, stop);
        int diff_hist;
        diff_hist = compute_diff_hist(hist, cpu_ref_hist, 256);
        if(diff_hist == 0){
            cout << "Histogram time GPU:" << msecTotal << "ms" << endl;
        }

        hipFree(histGPU);
    //}
    ///////////////////////////////////////////////////////

    //int* hist = new int[256]();
    int min,max;
    int temp = 0;

    for (int i = 0;i<255;i++){
        temp += hist[i];
        if(temp > size2*0.1){
            min = i;
            temp = 0;
            break;
        }
    }
    temp = 0;
    for (int i=255;i>=0;i--){
        temp += hist[i];
        if(temp > size2*0.1){
            max = i;
            temp = 0;
            break;
        }
    }
   
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    hipError_t error3 = hipMalloc((void**)&GPU_contrast, width*height*sizeof(unsigned char));

    if (error3 != hipSuccess){
        cout << "Error 3" << endl;
        if(error3 == hipErrorInvalidValue){
            cout<< "Invalid value" << endl;
        }
        if(error3 == cudaErrorInvalidSurface){
            cout<< "Invalid surface" << endl;
        }

    }

    dim3 grdBlkCon (128, 1,1);
    dim3 grdDimCon (ceil(size2/128),1,1);
    ContrastEnhancement<<<grdDimCon,grdBlkCon>>>(d_dst,GPU_contrast,width,height,min,max);

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal2, start, stop);
    cout << "Contrast enhancement time GPU:" << msecTotal2 << "ms" << endl;






    //cout << +h_contrast[(height-1)*width]<< " " << +h_contrast[(height-1)*width+1] << " " << +h_contrast[(height-2)*width] << " "  << +h_contrast[(height-2)*width+1] << endl;
   /* hipMalloc(&d_dst_2, (width*height)*sizeof(unsigned char));

    dim3 blkDimSmth (16, 16, 1);
    dim3 grdDimSmth ((width + 16-1)/BLOCKSIZE2, (height + BLOCKSIZE2-1)/BLOCKSIZE2, 1);
    hipDeviceSynchronize();
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    Smoothing<<<grdDimSmth,blkDimSmth>>>(GPU_contrast,d_dst_2, width, height);
    
    // add other three kernels here
    // clock starts -> copy data to gpu -> kernel1 -> kernel2->kernel3->kernel 4 ->copy result to cpu -> clock stops

    //wait until kernel finishes
    hipDeviceSynchronize();
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    //hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);
    
    //copy back the result to CPU
    //hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);
    unsigned char* temp5 = new unsigned char [width*height];
    
    hipMemcpy(temp5, d_dst_2, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    std::cout << "correct: " << +temp5[12132] << " " << +temp5[120] << " " << +temp5[8294400/2] << " " << +temp5[8294400/2+1] << endl;
*/
    cout << "Time of original smoothing function: " << msecTotal << endl;
    hipMalloc(&GPU_smoothing2, width*height*sizeof(unsigned char));




    //cout << +h_contrast[(height-1)*width]<< " " << +h_contrast[(height-1)*width+1] << " " << +h_contrast[(height-2)*width] << " "  << +h_contrast[(height-2)*width+1] << endl;

    int xthreads = 16;
    int ythreads = 16;
    hipDeviceSynchronize();
    dim3 blkDimSmthnew (xthreads, ythreads, 1);
    dim3 grdDimSmthnew (ceil(width/(xthreads-2)), ceil(height/(ythreads-2)), 1);
    //int maxdim = (xthreads+2)*(ythreads+2);
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    Smoothing_new<<<grdDimSmthnew,blkDimSmthnew>>>(GPU_contrast,GPU_smoothing2, width, height, xthreads, ythreads);
    hipDeviceSynchronize();
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    cout << "Time of optimised smoothing function: " << msecTotal << endl;

    if ( hipSuccess != hipGetLastError() )
    cout << "Error!\n";
    //wait until kernel finishes


    unsigned char* temp2 = new unsigned char[width*height];
    
    error3 = hipMemcpy(h_dst, GPU_smoothing2, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (error3 != hipSuccess){
        cout << "Error 3" << endl;
        if(error3 == hipErrorInvalidValue){
            cout<< "Invalid value" << endl;
        }
        if(error3 == cudaErrorInvalidSurface){
            cout<< "Invalid surface" << endl;
        }

    }

    //cout << "new: " << +temp2[1] << " " << +temp2[0] <<" " << +temp2[8294400/2] << " " << +temp2[8294400/2+1] << endl;

    
/*cout << "size " << size2 << endl;
int i=0;
    while(i < size2){
        if(temp2[i] != 0){
            cout << i << " integer " << +temp2[i] << endl;
            break;
        }
        i++;
    }*/
    //int res = compute_diff(h_dst,temp5,width*height);
    
  //  cout << res << " diff " << endl;


    hipFree(GPU_contrast);
    hipFree(GPU_smoothing);
    hipFree(GPU_smoothing2);
    //hipFree(histGPU);
    hipFree(d_src);


    hipFree(d_src);
    hipFree(d_dst);

    hipDeviceReset();
   // std::cout << "diff cpu and gpu " << res <<std::endl; // do not change this
    //std::cout <<"CPU processing time: " << gpu_time << " ms" <<std::endl; // do not change this
    //you need to save your final output, we need to measure the correctness of your program
    //read test.cpp to learn how to save a image
    //smoothing_gpu.save("smoothing_gpu.jpg"); 
  
    FILE * pFile;
    pFile = fopen ("gpu_out.txt","w");
    for (int i = 0; i < width; i++){
        for (int j = 0; j < height; j++){
            fprintf(pFile, "%d ", +h_dst[j*width+i]);
        }
        fprintf(pFile, "\n");
    }
    fclose(pFile);
    contrast.save("Con_GPU.jpg");
    //&h_dst = h_src;
    dst.save("file.jpg");
    //cout << +h_dst[height*width]<<endl;
    return 0;
}
