#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#define cimg_use_jpeg
#include "CImg.h"
#include <iostream>
#include "kernel.h"
#include "kernelcpu.h"
//#include "hip/hip_runtime_api.h"
#define BLOCKSIZE 128
#define BLOCKSIZE2 32
using namespace std;
using namespace cimg_library;


int compute_diff(unsigned char * res_cpu, unsigned char * res_gpu, unsigned long size){
  int res = 0;
  for(int i = 0;i < size; i++){
    res += res_cpu[i] - res_gpu[i];
  }
  return res;
}

int compute_diff_hist(int * res_cpu, int * res_gpu, unsigned long size){
    int res = 0;
    for(int i = 0;i < size; i++){
      res += res_cpu[i] - res_gpu[i];
    }
    return res;
  }


int main()
{
    //load image
    CImg<unsigned char> src("cat2.jpg"); // we use cat2.jpg to grade
    int width = src.width();
    int height = src.height();
    unsigned long size = src.size();
    unsigned long size2 = width*height;

    //hipDeviceProp_t deviceProp;
    //hipGetDeviceProperties(&deviceProp, dev);

    //create pointer to image
    unsigned char *h_src = src.data();
    
    CImg<unsigned char> dst(width, height, 1, 1);
    unsigned char *h_dst = dst.data();
    //Something contrast blabla
    CImg<unsigned char> contrast(width, height, 1, 1);
    unsigned char *h_contrast = contrast.data();

    // for contrast enhancemant
    CImg<unsigned char> smoothing_gpu(width, height, 1, 1);
    unsigned char *h_smoothing = smoothing_gpu.data();

    unsigned char *d_src;
    unsigned char *d_dst;

    unsigned char *GPU_contrast;
    unsigned char *GPU_smoothing;

    hipEvent_t start; // to record processing time
    hipEvent_t stop;
    float msecTotal,msecTotal2;
  


    std::cout << "Start CPU processing" << std::endl;
    // create and start timer

    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    unsigned char *cpu_ref = new unsigned char [width*height];
    int* cpu_ref_hist = new int[256];
    memset(cpu_ref_hist, 0, sizeof(int)*256);

    rgb2gray_cpu(h_src, cpu_ref, width, height);

    // hisgram cpu
    
    histgram_cpu(cpu_ref_hist, cpu_ref, width,  height);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    float cpu_time = msecTotal;

    std::cout <<"CPU processing time: " << cpu_time << " ms" <<std::endl;

    
    //std::cout << "Start GPU processing" << std::endl;
    // create and start timer
    //hipEventCreate(&start);
    //hipEventRecord(start, NULL); 

    hipMalloc((void**)&d_src, size);
    hipMalloc((void**)&d_dst, width*height*sizeof(unsigned char));

    hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice);

////////////////////////////////////////////////
/// RGB to gray ///

    //launch the kernel
    dim3 blkDim (BLOCKSIZE, 1,1);
    dim3 grdDim (ceil(size2/BLOCKSIZE), 1, 1);
    cout << (ceil(size2/BLOCKSIZE)) << " grid" << endl;
    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 

    rgb2gray<<<grdDim, blkDim>>>(d_src, d_dst, width, height);

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    cout << "RGB2Gray time GPU:" << msecTotal << "ms" << endl;

    hipDeviceSynchronize();
    hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);
    
//////////////////////////////////////////////////
    //for(int AmountOfHists = 16; AmountOfHists <= 16; AmountOfHists+= 4){
        int power = 3;
        int AmountOfHists = pow(2,power);
        int* hist = new int[256];
        int* histGPU = new int[256*AmountOfHists];
        // create and start timer
    ///////////////////////////////////////////////////////
        dim3 blkDim2 (512, 1,1);
        dim3 grdDim2 (ceil(size2/512),1, 1);


        bool temp3 = hipMalloc(&histGPU, 256*sizeof(int)*AmountOfHists) == hipSuccess;
        bool temp2 = hipMemset(histGPU, 0, 256*sizeof(int)*AmountOfHists) == hipSuccess;
        cout << "malloc is " << temp3 << " memset is " << temp2 <<endl;
        int mask = pow(2,(power))-1;
        hipEventCreate(&start);
        hipEventRecord(start, NULL); 
        histgram<<<grdDim2,blkDim2>>>(histGPU, d_dst, width , mask);
        hipEventCreate(&stop);
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&msecTotal2, start, stop);
        cout << "Histogram time GPU:" << msecTotal2 << "ms" << " for histnumber " << AmountOfHists << endl;

////////////////////////////////////////////////////////////////
        hipEventCreate(&start);
        hipEventRecord(start, NULL); 
        dim3 blkDim3 (256,1,1);
        for (int stride = AmountOfHists/2; stride>0; stride>>=1){
            dim3 grdDim3 (stride,1,1);
            histgram_summation<<<grdDim3, blkDim3>>>(histGPU, stride);
            hipDeviceSynchronize();
        }
        hipEventCreate(&stop);
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&msecTotal2, start, stop);
        cout << "Histogram summation time GPU:" << msecTotal2 << "ms" << " for histnumber " << AmountOfHists << endl;

        hipMemcpy(hist, histGPU, 256*sizeof(int),hipMemcpyDeviceToHost);
        cout << "CPU: " << cpu_ref_hist[0] << " GPU " << hist[0] << endl;
        /*for (int i = 0; i < 256; i++){
            for (int j = 1; j < 16; j++){
                hist[i] += hist[i+j*256];
            }
        }*/
        cout << "CPU: " << cpu_ref_hist[0] << " GPU " << hist[0] << endl;
        hipEventCreate(&stop);
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&msecTotal2, start, stop);
        int diff_hist;
        diff_hist = compute_diff_hist(hist, cpu_ref_hist, 256);
        if(diff_hist == 0){
            cout << "Histogram time GPU:" << msecTotal2 << "ms" << " for histnumber " << AmountOfHists << endl;
        }

        hipFree(histGPU);
    //}
    ///////////////////////////////////////////////////////

    //int* hist = new int[256]();
    int min,max;
    int temp = 0;

    for (int i = 0;i<255;i++){
        temp += hist[i];
        if(temp > size2*0.1){
            min = i;
            temp = 0;
            break;
        }
    }
    temp = 0;
    for (int i=255;i>=0;i--){
        temp += hist[i];
        if(temp > size2*0.1){
            max = i;
            temp = 0;
            break;
        }
    }
    //cout << "min " << min << " max " << max << endl;
    hipMalloc((void**)&GPU_contrast, width*height*sizeof(unsigned char));
    ContrastEnhancement<<<grdDim,blkDim>>>(d_dst,GPU_contrast,width,height,min,max);

    hipMemcpy(h_contrast,GPU_contrast, width*height, hipMemcpyDeviceToHost);
    cout << +h_contrast[(height-1)*width]<< " " << +h_contrast[(height-1)*width+1] << " " << +h_contrast[(height-2)*width] << " "  << +h_contrast[(height-2)*width+1] << endl;
    hipMalloc(&GPU_smoothing, width*height*sizeof(unsigned char));
    Smoothing<<<grdDim,blkDim>>>(GPU_contrast,d_dst, width, height);
    //cout << "Managed" << endl;
    
    // add other three kernels here
    // clock starts -> copy data to gpu -> kernel1 -> kernel2->kernel3->kernel 4 ->copy result to cpu -> clock stops

    //wait until kernel finishes
    hipDeviceSynchronize();
    //hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);
    
    //copy back the result to CPU
    //hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    //float gpu_time = msecTotal;
    

    
    int res = compute_diff(cpu_ref,h_dst,width*height);

    hipFree(GPU_contrast);
    hipFree(GPU_smoothing);
    //hipFree(histGPU);
    hipFree(d_src);


    hipFree(d_src);
    hipFree(d_dst);

    hipDeviceReset();
    std::cout << "diff cpu and gpu " << res <<std::endl; // do not change this
    //std::cout <<"CPU processing time: " << gpu_time << " ms" <<std::endl; // do not change this
    //you need to save your final output, we need to measure the correctness of your program
    //read test.cpp to learn how to save a image
    //smoothing_gpu.save("smoothing_gpu.jpg"); 
  
    FILE * pFile;
    pFile = fopen ("gpu_out.txt","w");
    for (int i = 0; i < width; i++){
        for (int j = 0; j < height; j++){
            fprintf(pFile, "%d ", +h_dst[j*width+i]);
        }
        fprintf(pFile, "\n");
    }
    fclose(pFile);
    contrast.save("Con_GPU.jpg");
    //&h_dst = h_src;
    dst.save("file.jpg");
    //cout << +h_dst[height*width]<<endl;
    return 0;
}
