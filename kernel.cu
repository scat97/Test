#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "kernel.h"


// implement your kernels
__global__ void rgb2gray(unsigned char * d_src, unsigned char * d_dst, int width, int height)
{
    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    //int pos_y = blockIdx.y * blockDim.y + threadIdx.y;

    /*if (pos_x >= width || pos_y >= height)
        return;
*/
    /*
     * CImg RGB channels are split, not interleaved.
     * (http://cimg.eu/reference/group__cimg__storage.html)
     */
    unsigned char r = d_src[pos_x];
    unsigned char g = d_src[height* width + pos_x];
    unsigned char b = d_src[height * 2 * width + pos_x];

    d_dst[pos_x] = (unsigned char)((float)(r + g + b) / 3.0f + 0.5);
    //unsigned char gray = _gray > 255 ? 255 : _gray;
}

__global__ void histgram(int* hist, unsigned char * gray, int width, int height){
    __shared__ int histshared[256];
    if(threadIdx.x < 256){
    histshared[threadIdx.x] = 0;
    }
    __syncthreads();
    int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned char loc = gray[pos_x];
    atomicAdd(&histshared[loc], 1);
    __syncthreads();
    if (threadIdx.x >255) return;
    if(histshared[threadIdx.x] == 0) return;
    
    atomicAdd(&hist[threadIdx.x+(blockIdx.x&height)*256], histshared[threadIdx.x]);
    


}
__global__ void histgram_summation(int* hist, int stride){
  //int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
  //int pos_y = blockIdx.y*blockDim.y+threadIdx.y;
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  hist[bid*256+tid]+=hist[(stride+bid)*256+tid];
}

__global__ void ContrastEnhancement(unsigned char*gray,unsigned char*res,int width, int height, int min, int max){
    int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
    if(pos_x >= width*height) return;

    int val = gray[pos_x];
    //int k = (pos_x * 4)%width;
    //res[pos_x] = static_cast<unsigned char>( 255 * (val-min)/(max-min));
    //int row = pos_x/width;
    //int x = pos_x-row*width;
    //res[pos_x] = (255);


    if(val > max) res[pos_x] =255;
    else if(val < min) res[pos_x] =0;
    else res[pos_x] = static_cast<unsigned char> (255 * (val-min)/(max-min));
    
    //uchar newval = 50;// static_cast<unsigned char>  (255 * (val-min)/(max-min));
    //surf2Dwrite(newval, surftest, x*4, row);
    
    /*if(val > max) {
      unsigned char maxval = 255;
      surf2Dwrite(maxval, surftest, x, row);
    }
    else if (val < min) {
      unsigned char minval = 0;
      surf2Dwrite(minval, surftest, x, row);}

    else {
      unsigned char newval = static_cast<unsigned char>  (255 * (val-min)/(max-min));
      surf2Dwrite(newval, surftest, x, row);
    }*/



}

__global__ void Smoothing(unsigned char*gray,unsigned char*res,int width, int height){
  int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
  int pos_y = blockIdx.y*blockDim.y+threadIdx.y;
  
  if ((pos_x >= width*height) || (pos_y >= height)) return;

  /*if ((pos_x == 0) && (pos_y == 0)) {
    res[0] = static_cast<unsigned char>((gray[0] + gray[1] +gray[width] + gray[width+1]) / 9.0); }
     // top left
  else if ((pos_x == (width-1))&&(pos_y == 0)){
    res[width-1] = static_cast<unsigned char>((gray[width-1] + gray[width-2] +  gray[width*2-1] + gray[width*2-2]) / 9.0) ; }
    //top right
  else if ((pos_x == (width-1)) && (pos_y == (height-1))){
     res[width*height-1] = static_cast<unsigned char>( (gray[height*width-1] + gray[height*width-2] + gray[(height-1)*width-1] + gray[(height-1)*width-2])/9.0) ; }//4bottom right}

  else if ((pos_x == 0) && (pos_y == (height-1))) {
    res[(height-1)*width] =  static_cast<unsigned char>((gray[(height-1)*width] + gray[(height-1)*width + 1] + gray[(height-2)*width] + gray[(height-2)*width+1]) / 9.0); }//4bottom left}

  else if(pos_y == 0){
    res[pos_x] =static_cast<unsigned char>( (gray[pos_x] + gray[pos_x+1] + gray[pos_x-1] + gray[ width+ pos_x-1] + gray[ width+ pos_x] + gray[ width+ pos_x+1]) / 9.0); // 6top row
  }
  else if(pos_y == (height-1)){
     res[pos_x+(height-1)*width] = static_cast<unsigned char>( (gray[(height-1)*width+pos_x] + gray[(height-1)*width+pos_x+1] + gray[(height-1)*width+pos_x-1] + gray[(height-2)*width+ pos_x-1] + gray[(height-2)*width+ pos_x] + gray[ (height-2)*width+ pos_x+1]) / 9.0);} // top row
  else if(pos_x == 0){
    res[width*pos_y] = static_cast<unsigned char>((gray[width*pos_y] + gray[width*pos_y+1] + gray[width*(pos_y-1)] + gray[width*(pos_y-1)+1] + gray[width*(pos_y+1)] + gray[width*(pos_y+1)+1]) / 9.0);}
  else if(pos_x == (width-1)){
    res[width*(pos_y+1)-1] = static_cast<unsigned char>((gray[width*(pos_y+1) - 1] + gray[width*(pos_y+1) - 1 -1] + gray[width*(pos_y) - 1] + gray[width*(pos_y) - 1 -1] + gray[width*(pos_y+2) - 1] +  gray[width*(pos_y+2) - 1 - 1]) / 9.0);}
  else {
    unsigned char val = static_cast<unsigned char>(((gray[pos_y * width + pos_x] + gray[pos_y * width + pos_x -1 ] + gray[pos_y * width + pos_x +1 ] + gray[(pos_y - 1)* width + pos_x  ] + gray[(pos_y + 1)* width + pos_x  ]  + gray[(pos_y-1) * width + pos_x-1] + gray[(pos_y-1) * width + pos_x+1] + gray[(pos_y+1) * width + pos_x-1] + gray[(pos_y+1) * width + pos_x+1] ) / 9.0 ));
    res[width*pos_y+pos_x] = val;}
*/
  if ((pos_x == 0) && (pos_y == 0)) {
    res[0] = static_cast<unsigned char>((gray[0] + gray[1] +gray[width] + gray[width+1]) / 4.0); }
     // top left
  else if ((pos_x == (width-1))&&(pos_y == 0)){
    res[width-1] = static_cast<unsigned char>((gray[width-1] + gray[width-2] +  gray[width*2-1] + gray[width*2-2]) / 4.0) ; }
    //top right
  else if ((pos_x == (width-1)) && (pos_y == (height-1))){
     res[width*height-1] = static_cast<unsigned char>( (gray[height*width-1] + gray[height*width-2] + gray[(height-1)*width-1] + gray[(height-1)*width-2])/4.0) ; }//bottom right}

  else if ((pos_x == 0) && (pos_y == (height-1))) {
    res[(height-1)*width] =  static_cast<unsigned char>((gray[(height-1)*width] + gray[(height-1)*width + 1] + gray[(height-2)*width] + gray[(height-2)*width+1]) / 4.0); }//bottom left}

  else if(pos_y == 0){
    res[pos_x] =static_cast<unsigned char>( (gray[pos_x] + gray[pos_x+1] + gray[pos_x-1] + gray[ width+ pos_x-1] + gray[ width+ pos_x] + gray[ width+ pos_x+1]) / 6.0); // top row
  }
  else if(pos_y == (height-1)){
     res[pos_x+(height-1)*width] = static_cast<unsigned char>( (gray[(height-1)*width+pos_x] + gray[(height-1)*width+pos_x+1] + gray[(height-1)*width+pos_x-1] + gray[(height-2)*width+ pos_x-1] + gray[(height-2)*width+ pos_x] + gray[ (height-2)*width+ pos_x+1]) / 6.0);} // top row
  else if(pos_x == 0){
    res[width*pos_y] = static_cast<unsigned char>((gray[width*pos_y] + gray[width*pos_y+1] + gray[width*(pos_y-1)] + gray[width*(pos_y-1)+1] + gray[width*(pos_y+1)] + gray[width*(pos_y+1)+1]) / 6.0);}
  else if(pos_x == (width-1)){
    res[width*(pos_y+1)-1] = static_cast<unsigned char>((gray[width*(pos_y+1) - 1] + gray[width*(pos_y+1) - 1 -1] + gray[width*(pos_y) - 1] + gray[width*(pos_y) - 1 -1] + gray[width*(pos_y+2) - 1] +  gray[width*(pos_y+2) - 1 - 1]) / 6.0);}
  else {
    unsigned char val = static_cast<unsigned char>(((gray[pos_y * width + pos_x] + gray[pos_y * width + pos_x -1 ] + gray[pos_y * width + pos_x +1 ] + gray[(pos_y - 1)* width + pos_x  ] + gray[(pos_y + 1)* width + pos_x  ]  + gray[(pos_y-1) * width + pos_x-1] + gray[(pos_y-1) * width + pos_x+1] + gray[(pos_y+1) * width + pos_x-1] + gray[(pos_y+1) * width + pos_x+1] ) / 9.0 ));
    res[width*pos_y+pos_x] = val;}
}

__global__ void Smoothing_new(unsigned char*gray,unsigned char *res,int width, int height, const int blwd, const int blht){

  int width2 = blockDim.x;
  int pos_x2 = threadIdx.x;
  int pos_y2 = threadIdx.y;
  __shared__ pixel tempout[18*18];

 tempout[pos_x2+width2*pos_y2].tleft = 0;
  tempout[pos_x2+width2*pos_y2].tmid = 0;
  tempout[pos_x2+width2*pos_y2].tright = 0;
  tempout[pos_x2+width2*pos_y2].mleft = 0;
  tempout[pos_x2+width2*pos_y2].mmid = 0;
  tempout[pos_x2+width2*pos_y2].mright = 0;
  tempout[pos_x2+width2*pos_y2].bleft = 0;
  tempout[pos_x2+width2*pos_y2].bmid = 0;
  tempout[pos_x2+width2*pos_y2].bright = 0;
  int pos_x = blockIdx.x*(blockDim.x-2)+threadIdx.x;
  int pos_y = blockIdx.y*(blockDim.y-2)+threadIdx.y;
  
  if ((pos_x >= width) || (pos_y >= height)) return;


    int val = static_cast<int>(gray[pos_x+pos_y*width]);
    
    tempout[pos_x2+width2*pos_y2].bright+=val;
    tempout[pos_x2+1+width2*pos_y2].bmid+=val;
    tempout[pos_x2+2+width2*pos_y2].bleft+=val;

    tempout[pos_x2+width2*(pos_y2+1)].mright+=val;
    tempout[pos_x2+1+width2*(pos_y2+1)].mmid+=val;
    tempout[pos_x2+2+width2*(pos_y2+1)].mleft+=val;

    tempout[pos_x2+width2*(pos_y2+2)].tright+=val;
    tempout[pos_x2+1+width2*(pos_y2+2)].tmid+=val;
    tempout[pos_x2+2+width2*(pos_y2+2)].tleft+=val;
    //

  if((threadIdx.x == 0) || (threadIdx.x == blockDim.x-1) || (threadIdx.y == 0) || (threadIdx.y == blockDim.y-1)){
    return;
  }
  //if((pos_x < 20) && (pos_y ==1)){printf("%d %d\n", pos_x, tempout[pos_x2+width2*pos_y2].mid );}
  __syncthreads();
  val = tempout[pos_x2+1+width2*(pos_y2+1)].tleft+tempout[pos_x2+1+width2*(pos_y2+1)].tmid+tempout[pos_x2+1+width2*(pos_y2+1)].tright+tempout[pos_x2+1+width2*(pos_y2+1)].mleft+tempout[pos_x2+1+width2*(pos_y2+1)].mmid+tempout[pos_x2+1+width2*(pos_y2+1)].mright+tempout[pos_x2+1+width2*(pos_y2+1)].bleft+tempout[pos_x2+1+width2*(pos_y2+1)].bmid+tempout[pos_x2+1+width2*(pos_y2+1)].bright;

  //res[pos_x+width*pos_y] = static_cast<unsigned char> ((tempout[pos_x2+1+width2*(pos_y2+1)].left+tempout[pos_x2+1+width2*(pos_y2+1)].mid+tempout[pos_x2+1+width2*(pos_y2+1)].right)/9.0);

  if((pos_x == 0)){
    if((pos_y == 0) || (pos_y == width-1)){
      res[pos_x+width*pos_y] = static_cast<unsigned char> (val/4.0);
    }
    else 
      res[pos_x+width*pos_y] = static_cast<unsigned char> (val/6.0);
  }
  else if(pos_x == width-1){
    if(pos_y == 0){
      res[pos_x+width*pos_y] = static_cast<unsigned char> (val/4.0);
    }
    else if(pos_y == width-1){
      res[pos_x+width*pos_y] = static_cast<unsigned char> (val/4.0);
    }
    else 
      res[pos_x+width*pos_y] = static_cast<unsigned char> (val/6.0);
  }
  else if(pos_y == 0){
    res[pos_x+width*pos_y] = static_cast<unsigned char> (val/6.0);
  }
  else if(pos_y == height-1){
    res[pos_x+width*pos_y] = static_cast<unsigned char> (val/6.0);
  }
  else res[pos_x+width*pos_y] = static_cast<unsigned char> (val/9.0);
  
  //unsigned char data1 = static_cast<unsigned char> (tex2D(texRef,pos_x,pos_y)*255);
  /*unsigned char data1;
  int mul = 4;
  surf2Dread(&data1,surftest, pos_x*mul, pos_y);
  
  unsigned char data2;
  unsigned char data3;
  unsigned char data4;
  unsigned char data5;
  unsigned char data6;
  unsigned char data7;
  unsigned char data8;
  unsigned char data9;
  int mul = 1;
  surf2Dread(&data1,surftest, pos_x*mul, pos_y);

  surf2Dread(&data2,surftest, (pos_x+1)*mul , (pos_y-1));
  surf2Dread(&data3,surftest, (pos_x-1)*mul, pos_y-1);
  surf2Dread(&data4,surftest, pos_x*mul , pos_y-1);
  surf2Dread(&data5,surftest, (pos_x-1)*mul, pos_y);
  surf2Dread(&data6,surftest, (pos_x+1)*mul, pos_y);
  surf2Dread(&data7,surftest, (pos_x)*mul, pos_y+1);
  surf2Dread(&data8,surftest, (pos_x+1)*mul, pos_y+1);
  surf2Dread(&data9,surftest, (pos_x-1)*mul, pos_y+1);*/
//res[pos_x+width*pos_y] = tex2D(texRef,pos_x,pos_y;//static_cast<unsigned char>((data1+data2+data3+data4+data5+data6+data7+data8+data9)/9.0);
  /*
  __shared__ int interresult[blockDim.x+2][blockDim.y+2];

  for(int i = -1; i < 2; i++){
    atomicAdd(&interresult[pos_x+i+1][(pos_y)],gray[pos_x+(pos_y)*width]);
    atomicAdd(&interresult[pos_x+i+1][(pos_y+2)],gray[pos_x+(pos_y)*width]);
    atomicAdd(&interresult[pos_x+i+1][(pos_y+1)],gray[pos_x+(pos_y)*width]);
  }

  __syncthreads();
  res[(pos_x+1)+(width+2)*(pos_y+1)] = interresult[pos_x+1][pos_y+1];
  if((threadIdx.x == 0) | (threadIdx.x = blockDim.x-1) | (threadIdx.y == 0) | (threadIdx.y == blockDim.y-1)){
    atomicAdd(&res[pos_x])
  }
*/
}

////////////////////////////////////////// CPU functions
void rgb2gray_cpu(unsigned char * d_src, unsigned char * d_dst, int width, int height){

  for (int i = 0; i < width ; i++){
    for (int j = 0; j < height ; j++){
      unsigned char r = d_src[j * width + i];
      unsigned char g = d_src[(height + j ) * width + i];
      unsigned char b = d_src[(height * 2 + j) * width + i];
      unsigned int _gray = (unsigned int)((float)(r + g + b) / 3.0f + 0.5);
      unsigned char gray = _gray > 255 ? 255 : _gray;
      d_dst[j * width + i] = gray;
    }
  }

}

void histgram_cpu(int* hist, unsigned char*gray,int width, int height){
  int size = width*height;

  for (int i=0;i<size;i++){
      unsigned char gray_val=gray[i];
      hist[gray_val]++;
  }
}
