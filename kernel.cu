#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "kernel.h"
// implement your kernels
__global__ void rgb2gray(unsigned char * d_src, unsigned char * d_dst, int width, int height)
{
    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    //int pos_y = blockIdx.y * blockDim.y + threadIdx.y;

    /*if (pos_x >= width || pos_y >= height)
        return;
*/
    /*
     * CImg RGB channels are split, not interleaved.
     * (http://cimg.eu/reference/group__cimg__storage.html)
     */
    unsigned char r = d_src[pos_x];
    unsigned char g = d_src[height* width + pos_x];
    unsigned char b = d_src[height * 2 * width + pos_x];

    d_dst[pos_x] = (unsigned char)((float)(r + g + b) / 3.0f + 0.5);
    //unsigned char gray = _gray > 255 ? 255 : _gray;
}

__global__ void histgram(int* hist, unsigned char * gray, int width, int height){
    __shared__ int histshared[256];
    if(threadIdx.x < 256){
    histshared[threadIdx.x] = 0;
    }
    __syncthreads();
    int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned char loc = gray[pos_x];
    atomicAdd(&histshared[loc], 1);
    __syncthreads();
    if (threadIdx.x >255) return;
    if(histshared[threadIdx.x] == 0) return;
    
    atomicAdd(&hist[threadIdx.x+(blockIdx.x&height)*256], histshared[threadIdx.x]);
    


}
__global__ void histgram_summation(int* hist, int stride){
  //int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
  //int pos_y = blockIdx.y*blockDim.y+threadIdx.y;
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  hist[bid*256+tid]+=hist[(stride+bid)*256+tid];
}

__global__ void ContrastEnhancement(unsigned char*gray,unsigned char*res,int width, int height, int min, int max){
    int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
    int pos_y = blockIdx.y*blockDim.y+threadIdx.y;

    if (pos_x >= width || pos_y >= height)
    return;

    int val = gray[pos_y*width+pos_x];
    if(val > max){
      res[pos_y*width+pos_x] = 255;
    }
    else if(val < min){
      res[pos_y*width+pos_x] = 0;
    }
    else{
      res[pos_y*width+pos_x] = static_cast<unsigned char>( 255 * (val-min)/(max-min));
    }


}

__global__ void Smoothing(unsigned char*gray,unsigned char*res,int width, int height){
  int pos_x = blockIdx.x*blockDim.x+threadIdx.x;
  int pos_y = blockIdx.y*blockDim.y+threadIdx.y;
  
  if (pos_x >= width || pos_y >= height) return;

  if ((pos_x == 0) && (pos_y == 0)) {
    res[0] = static_cast<unsigned char>((gray[0] + gray[1] +gray[width] + gray[width+1]) / 4.0); }
     // top left
  else if ((pos_x == (width-1))&&(pos_y == 0)){
    res[width-1] = static_cast<unsigned char>((gray[width-1] + gray[width-2] +  gray[width*2-1] + gray[width*2-2]) / 4.0) ; }
    //top right
  else if ((pos_x == (width-1)) && (pos_y == (height-1))){
     res[width*height-1] = static_cast<unsigned char>( (gray[height*width-1] + gray[height*width-2] + gray[(height-1)*width-1] + gray[(height-1)*width-2])/4.0) ; }//bottom right}

  else if ((pos_x == 0) && (pos_y == (height-1))) {
    res[(height-1)*width] =  static_cast<unsigned char>((gray[(height-1)*width] + gray[(height-1)*width + 1] + gray[(height-2)*width] + gray[(height-2)*width+1]) / 4.0); }//bottom left}

  else if(pos_y == 0){
    res[pos_x] =static_cast<unsigned char>( (gray[pos_x] + gray[pos_x+1] + gray[pos_x-1] + gray[ width+ pos_x-1] + gray[ width+ pos_x] + gray[ width+ pos_x+1]) / 6.0); // top row
  }
  else if(pos_y == (height-1)){
     res[pos_x+(height-1)*width] = static_cast<unsigned char>( (gray[(height-1)*width+pos_x] + gray[(height-1)*width+pos_x+1] + gray[(height-1)*width+pos_x-1] + gray[(height-2)*width+ pos_x-1] + gray[(height-2)*width+ pos_x] + gray[ (height-2)*width+ pos_x+1]) / 6.0);} // top row
  else if(pos_x == 0){
    res[width*pos_y] = static_cast<unsigned char>((gray[width*pos_y] + gray[width*pos_y+1] + gray[width*(pos_y-1)] + gray[width*(pos_y-1)+1] + gray[width*(pos_y+1)] + gray[width*(pos_y+1)+1]) / 6.0);}
  else if(pos_x == (width-1)){
    res[width*(pos_y+1)-1] = static_cast<unsigned char>((gray[width*(pos_y+1) - 1] + gray[width*(pos_y+1) - 1 -1] + gray[width*(pos_y) - 1] + gray[width*(pos_y) - 1 -1] + gray[width*(pos_y+2) - 1] +  gray[width*(pos_y+2) - 1 - 1]) / 6.0);}
  else {
    unsigned char val = static_cast<unsigned char>(((gray[pos_y * width + pos_x] + gray[pos_y * width + pos_x -1 ] + gray[pos_y * width + pos_x +1 ] + gray[(pos_y - 1)* width + pos_x  ] + gray[(pos_y + 1)* width + pos_x  ]  + gray[(pos_y-1) * width + pos_x-1] + gray[(pos_y-1) * width + pos_x+1] + gray[(pos_y+1) * width + pos_x-1] + gray[(pos_y+1) * width + pos_x+1] ) / 9.0 ));
    res[width*pos_y+pos_x] = val;}
}

////////////////////////////////////////// CPU functions
void rgb2gray_cpu(unsigned char * d_src, unsigned char * d_dst, int width, int height){

  for (int i = 0; i < width ; i++){
    for (int j = 0; j < height ; j++){
      unsigned char r = d_src[j * width + i];
      unsigned char g = d_src[(height + j ) * width + i];
      unsigned char b = d_src[(height * 2 + j) * width + i];
      unsigned int _gray = (unsigned int)((float)(r + g + b) / 3.0f + 0.5);
      unsigned char gray = _gray > 255 ? 255 : _gray;
      d_dst[j * width + i] = gray;
    }
  }

}

void histgram_cpu(int* hist, unsigned char*gray,int width, int height){
  int size = width*height;

  for (int i=0;i<size;i++){
      unsigned char gray_val=gray[i];
      hist[gray_val]++;
  }
}
